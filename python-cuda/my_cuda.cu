#include "hip/hip_runtime.h"
#include <Python.h>
#include <iostream>
#include "numpy/arrayobject.h"

using namespace std;

__global__
static void kernel(double *a, int N, double coeff) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index >= N)
		return;
	a[index] = index*coeff;
}

static void call_kernel(double *a, int N, double coeff) {
	int vector_size = 256;
	int gang = (N + vector_size-1)/vector_size;
	kernel<<<gang, vector_size>>>(a, N, coeff);
}

double *pyvector_to_Carrayptrs(PyArrayObject *arrayin)  {
    int i,n;
    
    n=arrayin->dimensions[0];
    return (double *) arrayin->data;  /* pointer to arrayin data as double */
}
/* ==== Check that PyArrayObject is a double (Float) type and a vector ==============
    return 1 if an error and raise exception */ 
int  not_doublevector(PyArrayObject *vec)  {
    if (vec->descr->type_num != NPY_DOUBLE || vec->nd != 1)  {
        PyErr_SetString(PyExc_ValueError,
            "In not_doublevector: array must be of type Float and 1 dimensional (n).");
        return 1;  }
    return 0;
}


static PyObject* helloworld(PyObject* self, PyObject* args)
{
    printf("Hello World\n");
    int N = 1000;
    double *a;
    hipMalloc((void**)&a, sizeof(double)*N);
    call_kernel(a, N, 1.0);
	double *ah = (double*) malloc(sizeof(double) * N);
	hipMemcpy(ah, a, sizeof(double)*N, hipMemcpyDefault);
	ah[10] = -1000;
	int max_count = 10;
	int counter = 0;
	for(int i=0;i<N;i++)
		if(ah[i] != i*1.0 && counter++ < max_count)
			cout << "Error in " << i << endl;

    return Py_None;
}


static PyObject *helloworld_input(PyObject *self, PyObject *args)
{
    PyArrayObject *vecin, *vecout;  // The python objects to be extracted from the args
    double *cin, *cout;             // The C vectors to be created to point to the 
                                    //   python vectors, cin and cout point to the row
                                    //   of vecin and vecout, respectively
    int n;
    double dfac;

    printf("====HELLO 0\n");
    
    /* Parse tuples separately since args will differ between C fcns */
    if (!PyArg_ParseTuple(args, "O!O!d", &PyArray_Type, &vecin,
        &PyArray_Type, &vecout, &dfac))  {printf("RIDI\n");return NULL;}
    if (NULL == vecin)  {printf("RIDI 2\n");return NULL;}
    if (NULL == vecout)  {printf("RIDI 3\n");return NULL;}

    printf("====HELLO 1\n");
    
    /* Check that objects are 'double' type and vectors
         Not needed if python wrapper function checks before call to this routine */
    if (not_doublevector(vecin)) return NULL;
    if (not_doublevector(vecout)) return NULL;
    
    /* Change contiguous arrays into C * arrays   */
    cin=pyvector_to_Carrayptrs(vecin);
    cout=pyvector_to_Carrayptrs(vecout);
    
    /* Get vector dimension. */
    n=vecin->dimensions[0];
    
    double *a;
    hipMalloc((void**)&a, sizeof(double)*n);
	hipMemcpy(a, cin, sizeof(double)*n, hipMemcpyDefault);
    call_kernel(a, n, dfac);
	hipMemcpy(cout, a, sizeof(double)*n, hipMemcpyDefault);
	hipFree(a);
        
    return Py_BuildValue("i", 1);
}


// Our Module's Function Definition struct
// We require this `NULL` to signal the end of our method
// definition
static PyMethodDef myMethods[] = {
    { "helloworld", helloworld, METH_NOARGS, "Prints Hello World" },
    { "helloworld_input", helloworld_input, METH_VARARGS},
    { NULL, NULL, 0, NULL }
};

// Initializes our module using our above struct
PyMODINIT_FUNC initgpuadder(void)
{
    Py_InitModule("gpuadder", myMethods);
	import_array();
}

